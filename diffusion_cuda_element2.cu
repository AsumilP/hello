#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


int NX;
int NY;
int BS;
float *data;
float *data_gpu;

/* in microseconds (us) */
long time_diff_us(struct timeval st, struct timeval et) {
    return (et.tv_sec-st.tv_sec)*1000000+(et.tv_usec-st.tv_usec);
}

void init(float *data) {
  int x, y;
  int cx = NX/2, cy = 0; /* center of ink */
  int rad = (NX+NY)/8; /* radius of ink */

  for(y = 0; y < NY; y++) {
    for(x = 0; x < NX; x++) {
      float v = 0.0;
      if (((x-cx)*(x-cx)+(y-cy)*(y-cy)) < rad*rad) {
	v = 1.0;
      }
      data[x+y*NX] = v;
      data[NX*NY+x+y*NX] = v;
    }
  }
  return;
}

/* Calculate for one time step */
/* Input: data[t%2], Output: data[(t+1)%2] */
__global__ void calc(float *data_gpu, int NX, int NY, int nt)
{
  int i, j, t;

  i = blockIdx.x* blockDim.x+ threadIdx.x;
  j = blockIdx.y* blockDim.y+ threadIdx.y;
  // j = blockIdx.x* blockDim.x+ threadIdx.x;
  // i = blockIdx.y* blockDim.y+ threadIdx.y;

  if (i == 0 || j == 0 || i >= NX-1 || j >= NY-1) return;

  for (t = 0; t < nt; t++) {
    int from = t%2;
    int to = (t+1)%2;

  data_gpu[NX*NY*to+i+j*NX] = 0.2 * (data_gpu[NX*NY*from+i+j*NX]
  			+ data_gpu[NX*NY*from+i-1+j*NX]
  			+ data_gpu[NX*NY*from+i+1+j*NX]
  			+ data_gpu[NX*NY*from+i+(j-1)*NX]
  			+ data_gpu[NX*NY*from+i+(j+1)*NX]);
  }
  return;
}

int  main(int argc, char *argv[])
{
  struct timeval tinb, tina, toutb, touta;
  int nt, i;
  hipError_t rc;
  FILE *fp;
  char filename[50];

  if (argc != 5){
    printf("Specify Grid_nx, Grid_ny, Timesteps_nt, BlockSize_BS\n");
  }else{
    NX = atoi(argv[1]);
    NY = atoi(argv[2]);
    nt = atoi(argv[3]);
    BS = atoi(argv[4]);
    printf("nx=%d, ny=%d, nt=%d, BS=%d \n", NX, NY, nt, BS);
  }

  data = (float *)malloc(sizeof(float)*2*NX*NY);
  init(data);
  rc = hipMalloc((void **)&data_gpu, sizeof(float)*2*NX*NY);
  if (rc != hipSuccess) {
    fprintf(stderr, "cudaMalloc, failed\n"); exit(1);
  }

  sprintf(filename, "./output/diffusion_cuda_s_gflop_nx%d_ny%d_nt%d_BS%d.dat", NX, NY, nt, BS);
  fp = fopen(filename,"w");

  for (i = 0; i < 5; i++) {
    hipDeviceSynchronize();
    gettimeofday(&tinb, NULL);
    rc = hipMemcpy(data_gpu, data, sizeof(float)*2*NX*NY, hipMemcpyHostToDevice);
    if (rc != hipSuccess) {
      fprintf(stderr, "cudaMemcpy, input failed\n"); exit(1);
    }
    hipDeviceSynchronize();
    gettimeofday(&tina, NULL);

    dim3 grid = dim3((NX+BS-1)/BS, ((NY+BS-1)/BS), 1);
    dim3 block = dim3(BS, BS, 1);
    calc<<<grid, block>>>(data_gpu, NX, NY, nt);

    hipDeviceSynchronize();
    gettimeofday(&toutb, NULL);
    hipDeviceSynchronize();
    rc = hipMemcpy(data, data_gpu, sizeof(float)*2*NX*NY, hipMemcpyDeviceToHost);
    if (rc != hipSuccess) {
      fprintf(stderr, "cudaMemcpy, output failed\n"); exit(1);
    }
    hipDeviceSynchronize();
    gettimeofday(&touta, NULL);
    {
        double us_in, us_out, us_calc;
        double gflops;
        int op_per_point = 5; // 4 add & 1 multiply per point

        us_in = time_diff_us(tinb, tina);
        us_in = us_in/1000000.0;
        us_calc = time_diff_us(tina, toutb);
        us_calc = us_calc/1000000.0;
        us_out = time_diff_us(toutb, touta);
        us_out = us_out/1000000.0;
        printf("Elapsed time for input: %.3lf sec\n", us_in);
        printf("Elapsed time for calc: %.3lf sec\n", us_calc);
        printf("Elapsed time for output: %.3lf sec\n", us_out);
        gflops = ((double)NX*NY*nt*op_per_point)/(us_in+us_calc+us_out)/1000000000.0;
        printf("Speed: %.3lf GFlops\n", gflops);
        fwrite(&us_in,8,1,fp);
        fwrite(&us_calc,8,1,fp);
        fwrite(&us_out,8,1,fp);
        fwrite(&gflops,8,1,fp);
    }
    init(data);
  }
  hipFree(data_gpu);
  free(data);
  fclose(fp);
  return 0;
}
